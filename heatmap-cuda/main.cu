#include "hip/hip_runtime.h"
﻿#include "main.h"

using namespace std;

__global__ void simulateRoundWithCuda(Heatmap* d_heatmap, int numberOfElements)
{
    // Calculate position in a flattened array
    int threadPositionFlat = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadPositionFlat < numberOfElements)
    {
        pair<int, int> coordinates = d_heatmap->getCoordinatesFromIndex(threadPositionFlat);
        d_heatmap->setValue(coordinates.first, coordinates.second, calculateFutureTemperature(*d_heatmap, coordinates.first, coordinates.second));
    }
}

int main(int argc, char **argv)
{
    if (argc < 3)
    {
        cout << "Error; not enough parameters specified, continuing with default parameters!" << endl;
        // return -2;
    }

    int fieldWidth = 20;
    int fieldHeight = 7;
    int numberOfRounds = 17;
    string hotspotFileName = (argc > 4) ? argv[4] : "hotspots.csv";
    string coordsFileName = (argc > 5) ? argv[5] : "";

    cout << "Reading arguments..." << endl;
    if (argc > 4)
    {
        fieldWidth = stoi(argv[1]);
        fieldHeight = stoi(argv[2]);
        numberOfRounds = stoi(argv[3]);
        hotspotFileName = argv[4];
    }

    if (argc > 5)
    {
        coordsFileName = argv[5];
    }

    Heatmap heatmap(fieldWidth, fieldHeight);
    Lifecycle lifecycles = Lifecycle();
    vector<pair<int, int>> coords;

    readData(hotspotFileName, lifecycles);
    readData(coordsFileName, coords);

    for (auto const &xy : coords)
    {
        cout << xy.first << ", " << xy.second << endl;
    }

    int numberOfElements = heatmap.getSize();

    int threadsPerBlock = 256;
    int blocksPerGrid = (numberOfElements + threadsPerBlock - 1) / threadsPerBlock;

    updateHotspots(heatmap, lifecycles, 0);
    heatmap.printFormattedOutputCout();

    // Create class storage on device and copy top level class
    Heatmap *d_heatmap;
    hipMalloc((void **)&d_heatmap, sizeof(Heatmap));
    hipMemcpy(d_heatmap, &heatmap, sizeof(Heatmap), hipMemcpyHostToDevice);
    // Make an allocated region on device for use by pointer in class
    double *d_data;
    hipMalloc((void **)&d_data, sizeof(double)*numberOfElements);
    hipMemcpy(d_data, heatmap.data, sizeof(double)*numberOfElements, hipMemcpyHostToDevice);
    int *d_width;
    hipMalloc((void **)&d_width, sizeof(int));
    hipMemcpy(d_width, &heatmap.width, sizeof(int), hipMemcpyHostToDevice);
    int *d_height;
    hipMalloc((void **)&d_height, sizeof(int));
    hipMemcpy(d_height, &heatmap.height, sizeof(int), hipMemcpyHostToDevice);
    // Copy pointer to allocated device storage to device class
    hipMemcpy(&(d_heatmap->data), &d_data, sizeof(double *), hipMemcpyHostToDevice);

    // Run Kernel
    simulateRoundWithCuda<<<threadsPerBlock, blocksPerGrid>>>(d_heatmap, numberOfElements);
    
    // Copy data to host
    hipDeviceSynchronize();
    hipMemcpy(&d_data, &(d_heatmap->data), sizeof(double)*numberOfElements, hipMemcpyDeviceToDevice);
    hipMemcpy(heatmap.data, d_data, sizeof(double)*numberOfElements, hipMemcpyDeviceToHost);
    // Cleanup
    hipFree(d_heatmap);
    hipFree(d_data);
    hipFree(d_width);
    hipFree(d_height);

    //for (int i = 0; i < numberOfRounds; i++)
    //{
    //    updateHotspots(heatmap, lifecycles, i);
    //    cout << "updateHotspots in Round " << i << "/" << numberOfRounds << endl;
    //    simulateRound(heatmap);
    //    cout << "simulateRound in Round " << i << "/" << endl;
    //    updateHotspots(heatmap, lifecycles, i+1);
    //    cout << "updateHotspots(i+1) in Round " << i << "/" << endl;
    //}

    if (coords.empty())
    {
        cout << "Print all coordinates." << endl;
        heatmap.printFormattedOutput();
    }
    else
    {
        cout << "Print selected coordinates." << endl;
        heatmap.printAtCoords(coords);
    }

    return 0;
}