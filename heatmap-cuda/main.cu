#include "hip/hip_runtime.h"
﻿#include "main.h"

using namespace std;

__global__ void simulateRoundWithCuda(Heatmap *heatmap, Heatmap *futureHeatmap, int numElements)
{
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    while (globalIdx < numElements)
    {
        pair<int, int> coordinates = heatmap->getCoordinatesFromIndex(globalIdx);
        futureHeatmap->setValue(coordinates, calculateFutureTemperature(*heatmap, coordinates.first, coordinates.second));

        globalIdx += blockDim.x * gridDim.x;
        __syncthreads();
    }
    heatmap = futureHeatmap;
}

int main(int argc, char **argv)
{
    if (argc < 3)
    {
        cout << "Error; not enough parameters specified, continuing with default parameters!" << endl;
        // return -2;
    }

    int fieldWidth = 20;
    int fieldHeight = 7;
    int numberOfRounds = 17;
    string hotspotFileName = (argc > 4) ? argv[4] : "hotspots.csv";
    string coordsFileName = (argc > 5) ? argv[5] : "";

    cout << "Reading arguments..." << endl;
    if (argc > 4)
    {
        fieldWidth = stoi(argv[1]);
        fieldHeight = stoi(argv[2]);
        numberOfRounds = stoi(argv[3]);
        hotspotFileName = argv[4];
    }

    if (argc > 5)
    {
        coordsFileName = argv[5];
    }

    Heatmap heatmap(fieldWidth, fieldHeight);
    Lifecycle lifecycles = Lifecycle();
    vector<pair<int, int>> coords;

    readData(hotspotFileName, lifecycles);
    readData(coordsFileName, coords);

    for (auto const &xy : coords)
    {
        cout << xy.first << ", " << xy.second << endl;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (heatmap.getSize() + threadsPerBlock - 1) / threadsPerBlock;

    // Copy data to device
    Heatmap futureHeatmap(heatmap.getWidth(), heatmap.getHeight());
    Heatmap *futureHeatmapPointer = &futureHeatmap;
    Heatmap *heatmapPointer = &heatmap;
    
    // Copy data to device
    hipMalloc(&futureHeatmapPointer, sizeof(futureHeatmap));
    hipMalloc(&heatmapPointer, sizeof(heatmap));

    for (int i = 0; i < numberOfRounds; i++)
    {
        updateHotspots(heatmap, lifecycles, i);
        cout << "Round " << i << ", before simulation: " << endl;
        heatmap.printFormattedOutputCout();
        cout << endl;
        hipMemcpy(&futureHeatmap, &futureHeatmap, sizeof(futureHeatmap), hipMemcpyHostToDevice);
        hipMemcpy(&heatmap, &heatmap, sizeof(heatmap), hipMemcpyHostToDevice);
        simulateRoundWithCuda<<<threadsPerBlock, blocksPerGrid>>>(&heatmap, &futureHeatmap, heatmap.getSize());
        hipDeviceSynchronize();
        hipMemcpy(&heatmap, &futureHeatmap, sizeof(heatmap), hipMemcpyDeviceToHost);
        cout << "Round " << i << ", after simulation" << endl;
        heatmap.printFormattedOutputCout();
        cout << endl;
        updateHotspots(heatmap, lifecycles, i + 1);
    }

    // Copy results back to device
    hipFree(&futureHeatmap);
    hipFree(&heatmap);

    if (coords.empty())
    {
        cout << "Print all coordinates." << endl;
        heatmap.printFormattedOutput();
    }
    else
    {
        cout << "Print selected coordinates." << endl;
        heatmap.printAtCoords(coords);
    }

    return 0;
}